#include "hip/hip_runtime.h"
﻿#include "kernels.cuh"

namespace GLoVe
{
	// Zeros an array
	__global__ void ZeroArrayKernel(
		const int length,
		float arr[]
	)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < length)
			arr[i] = 0.0;
	}

	// Zeros an array
	__global__ void ZeroArrayKernel(
		const int length,
		double arr[]
	)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < length)
			arr[i] = 0.0;
	}

	// Calcluates F_ij = f(X_ij)
	__global__ void CalcFKernel(
		const int C,
		const float X[],
		const float xmax, const float alpha,
		float F[]
	)
	{
		int c = blockIdx.x * blockDim.x + threadIdx.x;

		if (c < C)
			if (X[c] < xmax)
				F[c] = pow(X[c] / xmax, alpha);
			else
				F[c] = 1.0;
	}

	// Calculates L_ij = log(X_ij + 1)
	__global__ void CalcLKernel(
		const int C,
		const float X[],
		float L[]
	)
	{
		int c = blockIdx.x * blockDim.x + threadIdx.x;

		if (c < C)
			L[c] = log(X[c] + 1);
	}

	// Calculates M_ij = w_i.T @ w_j + b_i + b_j
	__global__ void CalcMKernel(
		const int C, const int P,
		const int rows[], const int cols[],
		const float W1[], const float W2[],
		const float b1[], const float b2[],
		float M[]
	)
	{
		int c = blockIdx.x * blockDim.x + threadIdx.x;

		if (c < C)
		{
			int i = rows[c];
			int j = cols[c];

			M[c] = 0.0;
			for (int p = 0; p < P; p++)
				M[c] += W1[i * P + p] * W2[j * P + p];
			M[c] += b1[i] + b2[j];
		}
	}

	// Calculates FoML = row_sum( F_ij * (M_ij - L_ij) )
	__global__ void CalcFoMLKernel(
		const int C,
		const int rows[], const float F[], const float M[], const float L[],
		float FoML[]
	)
	{
		int c = blockIdx.x * blockDim.x + threadIdx.x;

		if (c < C)
		{
			int k = rows[c];
			float val = F[c] * (M[c] - L[c]);
			atomicAdd(&FoML[k], val);
		}
	}

	// Calculates FoML = row_sum( F_ij * (M_ij - L_ij) )
	__global__ void CalcFoML2Kernel(
		const int C,
		const int rows[], const float F[], const float M[], const float L[],
		float FoML2[]
	)
	{
		int c = blockIdx.x * blockDim.x + threadIdx.x;

		if (c < C)
		{
			int k = rows[c];
			float val = F[c] * pow(M[c] - L[c], 2);
			atomicAdd(&FoML2[k], val);
		}
	}

	// Calculates J = sum( F_ij * (M_ij - L_ij)^2 )
	__global__ void CalcJKernel(
		const int V,
		const float FoML2[],
		float* J
	)
	{
		int k = blockIdx.x * blockDim.x + threadIdx.x;

		if (k < V)
			atomicAdd(J, FoML2[k]);
	}

	// Calculates DWJ
	__global__ void CalcDWJKernel(
		const int V, const int P,
		const float W1[], const float W2[],
		const float FoML[],
		float DW1J[], float DW2J[],
		float GW1[], float GW2[]
	)
	{
		int k = blockIdx.x * blockDim.x + threadIdx.x;
		int p = threadIdx.y;

		if (k < V && p < P)
		{
			int kp = k * P + p;

			// Calculate derivatives
			float dw1j = W1[kp] * FoML[k];
			float dw2j = W2[kp] * FoML[k];

			// Update derivatives
			DW1J[kp] = dw1j;
			DW2J[kp] = dw2j;

			//// Update sums of squared gradients
			GW1[kp] += pow(dw1j, 2);
			GW2[kp] += pow(dw2j, 2);
		}
	}

	// Calculates DbJ
	__global__ void CalcDbJKernel(
		const int V,
		const float FoML[],
		float DbJ[],
		float Gb[]
	)
	{
		int k = blockIdx.x * blockDim.x + threadIdx.x;

		if (k < V)
		{
			// Calculate derivative
			//float db = 2 * FoML[k];
			float db = FoML[k];

			// Update derivative
			DbJ[k] = db;

			//// Update sum of squared gradients
			Gb[k] += pow(db, 2);
		}
	}
	__global__ void UpdateVectorKernel(
		const int V, const int P,
		const float eta,
		const float DW1J[], const float DW2J[],
		const float GW1[], const float GW2[],
		float W1[], float W2[]
	)
	{
		int k = blockIdx.x * blockDim.x + threadIdx.x;
		int p = threadIdx.y;

		if (k < V && p < P)
		{
			int kp = k * P + p;
			W1[kp] -= (eta / sqrt(GW1[kp] + 10e-8)) * DW1J[kp];
			W2[kp] -= (eta / sqrt(GW2[kp] + 10e-8)) * DW2J[kp];
		}
	}
	__global__ void UpdateBiasKernel(
		const int V,
		const float eta,
		const float DbJ[],
		const float Gb[],
		float b1[], float b2[]
	)
	{
		int k = blockIdx.x * blockDim.x + threadIdx.x;

		if (k < V)
		{
			b1[k] -= (eta / sqrt(Gb[k] + 10e-8)) * DbJ[k];
			b2[k] -= (eta / sqrt(Gb[k] + 10e-8)) * DbJ[k];
		}
	}
}